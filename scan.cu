/*****************************************************************************
 * File:        scan.cu
 * Description: Implementing scan computation in different ways
 *              
 * Compile:     nvcc -o scan scan.cu -I..
 * Run:         ./scan
 *          [0] : Scan cpu
 *          [1] : Naive Parallel Scan
 *          [2] : Parallel Block Scan
 *          [3] : Parallel Block Scan Avoiding Bank Conflicts
 *          [4] : Parallel Large Array Scan
 *          [5] : Parallel Large Array Scan Avoiding Bank Conflicts
 *****************************************************************************/



#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <tuple>
#include <chrono>
#include <tuple>
#include <random>

#define NUM_BANKS 32
#define MAX_ELEMENTS_PER_BLOCK 2048
#define MAX_THREADS_PER_BLOCK 1024
#define LOG_NUM_BANKS 5
#define ZERO_BANK_CONFLICTS
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) (((n) >> LOG_NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif
#define MAX_SHARE_SIZE (MAX_ELEMENTS_PER_BLOCK + CONFLICT_FREE_OFFSET(MAX_ELEMENTS_PER_BLOCK - 1))
#define CUDA_CHECK(call)                                                                                               \
    do                                                                                                                 \
    {                                                                                                                  \
        hipError_t err = call;                                                                                        \
        if (err != hipSuccess)                                                                                        \
        {                                                                                                              \
            printf("CUDA Error: \n");                                                                                  \
            printf("    File:       %s\n", __FILE__);                                                                  \
            printf("    Line:       %d\n", __LINE__);                                                                  \
            printf("    Error Code: %d\n", err);                                                                       \
            printf("    Error Text: %s\n", hipGetErrorString(err));                                                   \
            exit(1);                                                                                                   \
        }                                                                                                              \
    } while (0)


//--------------------------------------------------------------------------------------------------------------------------//
//--------------------------------------------- utiles  ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------//

// 初始化一个整数数组 data，使其包含从 -10 到 100 的随机整数
void data_init(int *data, int N)
{
    //创建一个均匀分布的整数生成器，范围从 -10 到 100。这意味着生成的随机数在这个区间内是等概率的
    std::uniform_int_distribution<> int_generator(-10, 100); 
    // // 使用当前时间作为种子来初始化默认的随机数引擎 (rand_engine)。time(nullptr) 返回当前的系统时间（以秒为单位），确保每次调用程序时会有不同的初始值，从而生成不同的随机数序列。
    std::default_random_engine rand_engine(time(nullptr)); // 
    for (int i = 0; i < N; i++)
    {
        data[i] = int_generator(rand_engine);
    }
}

void results_check(int *a, int *b, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (a[i] != b[i])
        {
            printf("results_check fail\n");
            exit(1);
        }
    }
}

// 打印一个整数数组的内容
void print_int_arr(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("%d ", a[i]);
    }
    printf("\n");
}

int next_power_of_two(int x)
{
    int power = 1;
    while (power < x)
    {
        power *= 2;
    }
    return power;
}

// warm_up_kernel 和 warm_up 使用简单的kernel进行warm up
__global__ void warm_up_kernel(int *data)
{
    int tid = threadIdx.x;
    data[tid] += tid;
}

void warm_up()
{
    int N = 512;
    size_t arr_size = N * sizeof(int);
    int *data = (int *)malloc(arr_size);
    data_init(data, N);

    for (int i = 0; i < 10; i++)
    {
        int *d_data;
        CUDA_CHECK(hipMalloc(&d_data, arr_size));
        CUDA_CHECK(hipMemcpy(d_data, data, arr_size, hipMemcpyHostToDevice));

        warm_up_kernel<<<1, N>>>(d_data);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(data, d_data, arr_size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_data));
    }

    free(data);
}

// 定义了两个计时器类：TotalTimer 和 KernelTimer
class TotalTimer
{
  private:
    //分别记录计时的开始和结束时间点
    std::chrono::high_resolution_clock::time_point m_start_point, m_end_point;

  public:
    void start()
    {
        m_start_point = std::chrono::high_resolution_clock::now();
    };
    void end()
    {
        m_end_point = std::chrono::high_resolution_clock::now();
    };
    float cost()
    {
        // 通过计算结束时间与开始时间之间的差值来得到时间间隔，并将其转换为毫秒（std::milli）
        std::chrono::duration<float, std::milli> dur = m_end_point - m_start_point;
        return dur.count();
    };
};

// 专门用于测量 CUDA 内核的执行时间
class KernelTimer
{
  private:
    hipEvent_t m_start_event, m_end_event;

  public:
    KernelTimer()
    {
        CUDA_CHECK(hipEventCreate(&m_start_event));
        CUDA_CHECK(hipEventCreate(&m_end_event));
    };
    ~KernelTimer()
    {
        CUDA_CHECK(hipEventDestroy(m_start_event));
        CUDA_CHECK(hipEventDestroy(m_end_event));
    };
    void start()
    {
        CUDA_CHECK(hipEventRecord(m_start_event));
    };
    void end()
    {
        CUDA_CHECK(hipEventRecord(m_end_event));
        CUDA_CHECK(hipEventSynchronize(m_end_event));
    };
    float cost()
    {
        float kernel_cost;
        CUDA_CHECK(hipEventElapsedTime(&kernel_cost, m_start_event, m_end_event));
        return kernel_cost;
    };
};


//--------------------------------------------------------------------------------------------------------------------------//
//--------------------------------------------- scan  ----------------------------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------------//

//----------------------------
//------- scan_cpu -----------
//----------------------------
float scan_cpu(int *data, int *prefix_sum, int N)
{
    TotalTimer total_timer;
    total_timer.start();

    prefix_sum[0] = 0;
    for (int i = 1; i < N; i++)
    {
        prefix_sum[i] = prefix_sum[i - 1] + data[i - 1];
    }

    total_timer.end();
    return total_timer.cost();
}


//-----------------------------------
//------- native_scan -----------
//-----------------------------------
__global__ void native_scan_kernel(int *data, int *prefix_sum, int N)
{
    prefix_sum[0] = 0;
    for (int i = 1; i < N; i++)
    {
        prefix_sum[i] = prefix_sum[i - 1] + data[i - 1];
    }
}

std::tuple<float, float> native_scan_gpu(int *data, int *prefix_sum, int N)
{
    TotalTimer total_timer;
    total_timer.start();

    int *d_data, *d_prefix_sum;
    size_t arr_size = N * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_data, arr_size));
    CUDA_CHECK(hipMalloc(&d_prefix_sum, arr_size));
    CUDA_CHECK(hipMemcpy(d_data, data, arr_size, hipMemcpyHostToDevice));

    KernelTimer kernel_timer;
    kernel_timer.start();

    // 运行kernel
    native_scan_kernel<<<1, 1>>>(d_data, d_prefix_sum, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    kernel_timer.end();
    float kernel_cost = kernel_timer.cost();

    CUDA_CHECK(hipMemcpy(prefix_sum, d_prefix_sum, arr_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_prefix_sum));

    total_timer.end();
    float total_cost = total_timer.cost();

    return {total_cost, kernel_cost};
}

//---------------------------------------
//------- parallel_block_scan -----------
//---------------------------------------
__global__ void parallel_block_scan_kernel(int *data, int *prefix_sum, int N)
{
    extern __shared__ int tmp[];
    int tid = threadIdx.x;
    int leaf_num = blockDim.x * 2; // equals to length of tmp

    // 单线程块加载多数据块
    tmp[tid * 2] = tid * 2 < N ? data[tid * 2] : 0;
    tmp[tid * 2 + 1] = tid * 2 + 1 < N ? data[tid * 2 + 1] : 0;
    __syncthreads();

    // 并行扫描第一阶段：reduce规约
    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    // 开扫描，将最后一个元素记录为0
    if (tid == 0)
    {
        tmp[leaf_num - 1] = 0;
    }
    __syncthreads();

    // 并行扫描第二阶段：down sweep
    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            int v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    // 并行扫描结束将共享内存结果存回全局内存
    if (tid * 2 < N)
    {
        prefix_sum[tid * 2] = tmp[tid * 2];
    }
    if (tid * 2 + 1 < N)
    {
        prefix_sum[tid * 2 + 1] = tmp[tid * 2 + 1];
    }
}

// Bank conflict avoidance optimization (BCAO)
__global__ void parallel_block_scan_bcao_kernel(int *data, int *prefix_sum, int N)
{
    extern __shared__ int tmp[];
    int tid = threadIdx.x;
    int leaf_num = blockDim.x * 2; // not equals to length of tmp

    int ai = tid;
    int bi = tid + (leaf_num >> 1);
    int offset_ai = CONFLICT_FREE_OFFSET(ai);
    int offset_bi = CONFLICT_FREE_OFFSET(bi);

    tmp[ai + offset_ai] = ai < N ? data[ai] : 0;
    tmp[bi + offset_bi] = bi < N ? data[bi] : 0;
    __syncthreads();

    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (tid == 0)
    {
        tmp[leaf_num - 1 + CONFLICT_FREE_OFFSET(leaf_num - 1)] = 0;
    }
    __syncthreads();

    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    if (ai < N)
    {
        prefix_sum[ai] = tmp[ai + offset_ai];
    }
    if (bi < N)
    {
        prefix_sum[bi] = tmp[bi + offset_bi];
    }
}

std::tuple<float, float> parallel_block_scan_gpu(int *data, int *prefix_sum, int N, bool bcao)
{
    TotalTimer total_timer;
    total_timer.start();

    int *d_data, *d_prefix_sum;
    size_t arr_size = N * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_data, arr_size));
    CUDA_CHECK(hipMalloc(&d_prefix_sum, arr_size));
    CUDA_CHECK(hipMemcpy(d_data, data, arr_size, hipMemcpyHostToDevice));

    KernelTimer kernel_timer;
    kernel_timer.start();

    int padding_N = next_power_of_two(N);
    if (bcao)
    {
        int share_mem_size = (padding_N + CONFLICT_FREE_OFFSET(padding_N - 1)) * sizeof(int);
        parallel_block_scan_bcao_kernel<<<1, padding_N / 2, share_mem_size>>>(d_data, d_prefix_sum, N);
    }
    else
    {
        int share_mem_size = padding_N * sizeof(int);
        parallel_block_scan_kernel<<<1, padding_N / 2, share_mem_size>>>(d_data, d_prefix_sum, N);
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    kernel_timer.end();
    float kernel_cost = kernel_timer.cost();

    CUDA_CHECK(hipMemcpy(prefix_sum, d_prefix_sum, arr_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_prefix_sum));

    total_timer.end();
    float total_cost = total_timer.cost();

    return {total_cost, kernel_cost};
}


//--------------------------------------------
//------- Arrays of Arbitrary Size -----------
//--------------------------------------------
__global__ void parallel_large_scan_kernel(int *data, int *prefix_sum, int N, int *sums)
{
    __shared__ int tmp[MAX_ELEMENTS_PER_BLOCK];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int leaf_num = MAX_ELEMENTS_PER_BLOCK;

    tmp[tid * 2] = tid * 2 + block_offset < N ? data[tid * 2 + block_offset] : 0;
    tmp[tid * 2 + 1] = tid * 2 + 1 + block_offset < N ? data[tid * 2 + 1 + block_offset] : 0;
    __syncthreads();

    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (tid == 0)
    {
        sums[bid] = tmp[leaf_num - 1];
        tmp[leaf_num - 1] = 0;
    }
    __syncthreads();

    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            int v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    if (tid * 2 + block_offset < N)
    {
        prefix_sum[tid * 2 + block_offset] = tmp[tid * 2];
    }
    if (tid * 2 + 1 + block_offset < N)
    {
        prefix_sum[tid * 2 + 1 + block_offset] = tmp[tid * 2 + 1];
    }
}

__global__ void parallel_large_scan_bcao_kernel(int *data, int *prefix_sum, int N, int *sums)
{
    __shared__ int tmp[MAX_SHARE_SIZE];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int leaf_num = MAX_ELEMENTS_PER_BLOCK;

    int ai = tid;
    int bi = tid + (leaf_num >> 1);
    int offset_ai = CONFLICT_FREE_OFFSET(ai);
    int offset_bi = CONFLICT_FREE_OFFSET(bi);

    tmp[ai + offset_ai] = ai + block_offset < N ? data[ai + block_offset] : 0;
    tmp[bi + offset_bi] = bi + block_offset < N ? data[bi + block_offset] : 0;
    __syncthreads();

    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (tid == 0)
    {
        int last_idx = leaf_num - 1 + CONFLICT_FREE_OFFSET(leaf_num - 1);
        sums[bid] = tmp[last_idx];
        tmp[last_idx] = 0;
    }
    __syncthreads();

    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    if (ai + block_offset < N)
    {
        prefix_sum[ai + block_offset] = tmp[ai + offset_ai];
    }
    if (bi + block_offset < N)
    {
        prefix_sum[bi + block_offset] = tmp[bi + offset_bi];
    }
}

__global__ void add_kernel(int *prefix_sum, int *valus, int N)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int ai = tid + block_offset;
    int bi = tid + (MAX_ELEMENTS_PER_BLOCK >> 1) + block_offset;

    if (ai < N)
    {
        prefix_sum[ai] += valus[bid];
    }
    if (bi < N)
    {
        prefix_sum[bi] += valus[bid];
    }
}

void recursive_scan(int *d_data, int *d_prefix_sum, int N, bool bcao)
{
    int block_num = N / MAX_ELEMENTS_PER_BLOCK;
    if (N % MAX_ELEMENTS_PER_BLOCK != 0)
    {
        block_num += 1;
    }
    int *d_sums, *d_sums_prefix_sum;
    CUDA_CHECK(hipMalloc(&d_sums, block_num * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sums_prefix_sum, block_num * sizeof(int)));

    if (bcao)
    {
        parallel_large_scan_bcao_kernel<<<block_num, MAX_THREADS_PER_BLOCK>>>(d_data, d_prefix_sum, N, d_sums);
    }
    else
    {
        parallel_large_scan_kernel<<<block_num, MAX_THREADS_PER_BLOCK>>>(d_data, d_prefix_sum, N, d_sums);
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    if (block_num != 1)
    {
        recursive_scan(d_sums, d_sums_prefix_sum, block_num, bcao);
        add_kernel<<<block_num, MAX_THREADS_PER_BLOCK>>>(d_prefix_sum, d_sums_prefix_sum, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    CUDA_CHECK(hipFree(d_sums));
    CUDA_CHECK(hipFree(d_sums_prefix_sum));
}

std::tuple<float, float> parallel_large_scan_gpu(int *data, int *prefix_sum, int N, bool bcao)
{
    TotalTimer total_timer;
    total_timer.start();

    int *d_data, *d_prefix_sum;
    size_t arr_size = N * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_data, arr_size));
    CUDA_CHECK(hipMalloc(&d_prefix_sum, arr_size));
    CUDA_CHECK(hipMemcpy(d_data, data, arr_size, hipMemcpyHostToDevice));

    KernelTimer kernel_timer;
    kernel_timer.start();

    recursive_scan(d_data, d_prefix_sum, N, bcao);

    kernel_timer.end();
    float kernel_cost = kernel_timer.cost();

    CUDA_CHECK(hipMemcpy(prefix_sum, d_prefix_sum, arr_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_prefix_sum));

    total_timer.end();
    float total_cost = total_timer.cost();

    return {total_cost, kernel_cost};
}


//--------------------------------------------- main  ----------------------------------------------------------------------

int main(int argc, char **argv)
{
    warm_up();
    int nums[] = {1000, 2048, 100000, 10000000};
    int len = sizeof(nums) / sizeof(int);
    for (int i = 0; i < len; i++)
    {
        int N = nums[i];
        size_t arr_size = N * sizeof(int);
        int *data = (int *)malloc(arr_size);
        int *prefix_sum_cpu = (int *)malloc(arr_size);
        int *prefix_sum_gpu = (int *)malloc(arr_size);
        float total_cost, kernel_cost;
        data_init(data, N);
        printf("-------------------------- N = %d --------------------------\n", N);

        // ------ scan_cpu --------
        total_cost = scan_cpu(data, prefix_sum_cpu, N);
        printf("%35s - total: %10.5f ms\n", "scan_cpu", total_cost);

        // ------ native_scan --------
        std::tie(total_cost, kernel_cost) = native_scan_gpu(data, prefix_sum_gpu, N);
        results_check(prefix_sum_cpu, prefix_sum_gpu, N);
        printf("%35s - total: %10.5f ms    kernel: %10.5f ms\n", "native_scan_gpu", total_cost, kernel_cost);

        // 
        if (N <= MAX_ELEMENTS_PER_BLOCK)
        {
            std::tie(total_cost, kernel_cost) = parallel_block_scan_gpu(data, prefix_sum_gpu, N, false);
            results_check(prefix_sum_cpu, prefix_sum_gpu, N);
            printf("%35s - total: %10.5f ms    kernel: %10.5f ms\n", "parallel_block_scan_gpu", total_cost,
                   kernel_cost);

            std::tie(total_cost, kernel_cost) = parallel_block_scan_gpu(data, prefix_sum_gpu, N, true);
            results_check(prefix_sum_cpu, prefix_sum_gpu, N);
            printf("%35s - total: %10.5f ms    kernel: %10.5f ms\n", "parallel_block_scan_gpu with bcao", total_cost,
                   kernel_cost);
        }

        // 5. Arrays of Arbitrary Size scan
        std::tie(total_cost, kernel_cost) = parallel_large_scan_gpu(data, prefix_sum_gpu, N, false);
        results_check(prefix_sum_cpu, prefix_sum_gpu, N);
        printf("%35s - total: %10.5f ms    kernel: %10.5f ms\n", "parallel_large_scan_gpu", total_cost, kernel_cost);

        // 5. Arrays of Arbitrary Size scan without bank Conflicts
        std::tie(total_cost, kernel_cost) = parallel_large_scan_gpu(data, prefix_sum_gpu, N, true);
        results_check(prefix_sum_cpu, prefix_sum_gpu, N);
        printf("%35s - total: %10.5f ms    kernel: %10.5f ms\n", "parallel_large_scan_gpu with bcao", total_cost,
               kernel_cost);

        free(data);
        free(prefix_sum_cpu);
        free(prefix_sum_gpu);
        printf("\n");
    }
}